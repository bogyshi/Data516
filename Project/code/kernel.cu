#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>
#include <limits>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <algorithm>
#include <iterator>

#include <hip/hip_runtime.h>         // helper functions for CUDA error check

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif


#define SIZE 1024
#define SIZE2 89478484 

const int maximumSize = 1024 * 1024 * 1.5;
const int parameterSize = 48;
const int numCols = parameterSize;
//const int numRows = 10000;
const int numFiles = 168;
const int numRows = 1891422; // 1903771 - 12349

const int totalSize = 1024 * 1024 * 1;
const int totalCols = 256;
const int truSize = numRows;
const float LR = 10;
const int maxIters = 200;
const int checkPoint = maxIters/2;
const int maxAddress = numRows * numCols;

void readInData(float* data)
{
	std::ifstream rf("B:\\Github2\\Data512\\finalProject\\data\\inData2.bin", std::ios::in | std::ios::binary);
	rf.read((char*)&data[0], sizeof(float)*numRows*numCols);
	rf.close();
}

void readOutData(float * outputs)
{
	std::ifstream rf("B:\\Github2\\Data512\\finalProject\\data\\outData.bin", std::ios::in | std::ios::binary);
	rf.read((char*)&outputs[0], sizeof(int) * numRows);
	rf.close();
}

bool isNumber(const std::string& s) //https://stackoverflow.com/questions/29169153/how-do-i-verify-a-string-is-valid-double-even-if-it-has-a-point-in-it
{
	char* end = 0;
	double val = strtod(s.c_str(), &end);
	return end != s.c_str() && *end == '\0' && val != HUGE_VAL;
}

void setOutputs(float * output)
{
	std::fstream fin,badRows;
	std::string pathName = "B:\\Github2\\Data512\\finalProject\\data\\outputs.csv";
	badRows.open("B:\\Github2\\Data512\\finalProject\\data\\badRows.txt", std::fstream::in);
	fin.open(pathName, std::fstream::in);
	std::string v;
	std::vector<int> skipIndexes;
	while (badRows >> v)
	{
		skipIndexes.push_back(atoi(v.c_str()));
	}
	std::string result, line, temp, word;
	char* buffer = new char[3];
	int counter = 0;
	int colCounter = 0;
	bool skipCond;;

	while (fin >> temp)
	{
		std::stringstream s(temp);
		colCounter = 0;
		skipCond = std::find(std::begin(skipIndexes), std::end(skipIndexes), counter) == std::end(skipIndexes);
		while (std::getline(s, word, ','))
		{
			if (colCounter == 1 && skipCond)
			{
				output[counter] = atoi(word.c_str());
			}
			colCounter++;
		}
		counter++;
	}
}

void getDataDim()
{
	std::fstream fin;
	// Open an existing file 
	int counter = 1;
	int numRows = 0;
	std::string pathName = "B:\\Github2\\Data512\\finalProject\\data\\pivotData\\block";
	std::string result, line, temp, word;
	char* buffer = new char[3];
	result = pathName + "blockSpillsep.csv";
	fin.open(result, std::fstream::in);

	bool isgood = fin.good();
	bool begFile = false;
	int colCounter=0;
	int numBadRows = 0;
	std::vector<int> badRows;
	while (fin >> temp) {
		numRows++;
	}
	fin.close();

	while (counter < numFiles)
	{
		sprintf(buffer, "%d", counter);
		result = pathName + buffer + "sep.csv";
		fin.open(result, std::fstream::in);
		bool isgood = fin.good();
		begFile = true;
		while (fin >> temp) {
			std::stringstream s(temp);
			colCounter = 0;
			while (std::getline(s, word, ',')) {
				if (begFile)
				{
					break;
				}
				else if (colCounter>1 && (word.empty() || !isNumber(word))) //https://stackoverflow.com/questions/4654636/how-to-determine-if-a-string-is-a-number-with-c
				{
					badRows.push_back(numRows);
					numBadRows++;
					break;
				}
				colCounter++;
			}
			if (begFile == true)
			{
				begFile = false;
			}
			else
			{
				numRows++;
			}
		}
		fin.close();
		printf("%s\n", result.c_str());

		counter++;
	}
	std::ofstream fout("B:\\Github2\\Data512\\finalProject\\data\\badRows.txt");
	for (int x : badRows)
	{
		fout << x << "\n";
	}

	printf("Num total rows: %d\n", numRows);
	printf("Num bad rows: %d\n", badRows.size());
	printf("Num good rows: %d\n", numRows-numBadRows);


}
void read_record(std::vector<std::string> header, float * data)
{

	std::ifstream skipRows;
	skipRows.open("B:\\Github2\\Data512\\finalProject\\data\\badRows.txt", std::fstream::in);
	std::string v;
	std::vector<int> skipIndexes;
	while (skipRows>>v)
	{
		skipIndexes.push_back(atoi(v.c_str()));
	}

	// File pointer 
	std::fstream fin;

	// Open an existing file 
	int counter = 1;
	std::string pathName = "B:\\Github2\\Data512\\finalProject\\data\\pivotData\\block";
	std::string result,line,temp,word;
	char * buffer = new char[3];
	bool haveHeader = false;
	bool begOfFile = false;
	int i = 0;
	int whatRow=0;
	bool skipCond;
	int colCounter = 0;
	int address = 0;
	while (counter < numFiles)
	{
		sprintf(buffer, "%d", counter);
		result = pathName + buffer + "sep.csv";
		fin.open(result, std::fstream::in);
		begOfFile = true;
		while (fin >> temp) 
		{
			// used for breaking words 
			std::stringstream s(temp);
			// read every column data of a row and 
			// store it in a string variable, 'word'
			colCounter = 0;
			skipCond = std::find(std::begin(skipIndexes), std::end(skipIndexes), whatRow) == std::end(skipIndexes);
			if (skipCond)
			{
				while (std::getline(s, word, ',')) 
				{
					if (begOfFile == true)
					{
						if (haveHeader == false) {
							header.push_back(word);
						}
					}
					else if(colCounter>1)
					{
						address = i + (colCounter - 2) * numRows;
						data[address] = stof(word);
					}
					colCounter++;
				}
			}
			


			if (haveHeader == false)
			{
				haveHeader = true;
			}
			if (begOfFile == true)
			{
				begOfFile = false;
			}
			else
			{
				if(skipCond)                                                                          
					i++;
				whatRow++;
			}
		}
		fin.close();
		printf("%s\n",result.c_str());
		counter++;
	}
	//std::free(buffer);

	//roll2 = stoi(row[0]);

}

__global__ void vectorAdd2(int* a, int* b, int* c, int n)
{
	//int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int iw = threadIdx.x;
	int j = blockDim.x;
	int z = blockIdx.x;
	int iy = threadIdx.y;
	int jy = blockDim.y;
	int zy = blockIdx.y;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
	{
		c[i] = (a[i] + b[i]) % 1024;
	}
}

//hipMallocManaged(&b, truSize * sizeof(int));
//hipMallocManaged(&c, truSize * sizeof(int));


__global__ void logRegression(float * outs, float * ins, float * theta,float * weightChanges)
{

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = 0;
	double total = 0.0;
	//int addr = (numRows * j) + i;
	if (i < (numRows))
	{
		for (j = 0; j < numCols; j++)
		{
			total += theta[j] * ins[j*numRows+i];
		}
		float l = outs[i] - (1.0 / (1 + exp(-1 * total)));
		for (j = 0; j < numCols; j++)
		{
			float toadd = ((ins[(numRows * j) + i]) * l) / numRows;
			atomicAdd(&weightChanges[j], toadd);
		}
	}

}

__global__ void vectorAdd(int* cols, int numCols, int n)
{
	//int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int iw = threadIdx.x;
	//int j = blockDim.x;
	//int z = blockIdx.x;
	//int iy = threadIdx.y;
	//int jy = blockDim.y;
	//int zy = blockIdx.y;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = 0;
	if (i < n)
	{
		for (j = 1; j < numCols; j++)
		{
			cols[i] += cols[j * truSize + i];
		}
	}
}

int main()
{

	
	float * data = new float[numRows*numCols]; 
	float* output = new float[numRows];
	float * parameters = new float[numCols];
	float* toAdjust = new float[numCols];
	int i = 0;
	int j = 0;
	float insertVal = 0.0;
	float tochange = 0.0;
	int numIters = 0;
	float totalMilliseconds = 0;
	float milliseconds = 0;
	bool debug = false;
	int NT;
	int numThreadsArr[9] = {1024,512,256, 128, 64, 32,16,8,4};
	int toAvgPrf[9] = { 0,0,0,0,0,0,0,0,0};
	bool printRes = false;
	int roundCounter = 0;
	int whichAmt = 0;
	int numRounds = 20;
	hipError_t somtin;
	std::vector<std::string> header;


	//getDataDim(); // only need to do this once 1903771 - 12349;

	//34847,34961
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMallocManaged(&parameters, sizeof(float) * numCols);
	hipMallocManaged(&toAdjust, sizeof(double) * numCols);
	hipMallocManaged(&output, sizeof(float) * numRows);
	hipMallocManaged(&data, sizeof(float)*numRows*numCols);
	/*
	// we are missing some data at the end, but we will forget about it for now
	read_record(header, data);
	std::ofstream mydata2("B:\\Github2\\Data512\\finalProject\\data\\inData2.bin", std::ios::out | std::ios::binary);
	for (i = 0; i < 10; i++)
	{
		printf("last 10 vals of data from file: pos %d = %f", 10 - i, data[(numRows * numCols - 1) - i]);
	}
	mydata2.write((char*)&data[0], sizeof(float) * numRows * numCols);
	mydata2.close();
	*/
	readInData(data);
	for (i = 0; i < 10; i++)
	{
		printf("last 10 vals of data after write: pos %d = %f", 10 - i, data[(numRows * numCols - 1) - i]);
	}
	/*
	setOutputs(output);
	std::ofstream myOutdata2("B:\\Github2\\Data512\\finalProject\\data\\outData.bin", std::ios::out | std::ios::binary);
	myOutdata2.write((char*)&output[0], sizeof(int) * numRows);
	myOutdata2.close();
	*/
	readOutData(output);
	while (roundCounter < numRounds)
	{
		whichAmt = 0;
		for (int numThreadsFA : numThreadsArr)
		{
			NT = numThreadsFA;
			dim3 numThreads(NT);
			dim3 gridDim((numRows * numCols / NT) + 1, 1);

			// to view array contents, do "arrayName,numView" e.g. data,1
			for (i = 0; i < numCols; i++)
			{
				parameters[i] = -0.01;
				toAdjust[i] = 0.0;
			}

			numIters = 0;
			totalMilliseconds = 0;
			while (numIters < maxIters)
			{
				hipEventRecord(start);
				logRegression << <gridDim, numThreads >> > (output, data, parameters, toAdjust);
				hipEventRecord(stop);

				somtin = hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);
				totalMilliseconds += milliseconds;
				for (i = 0; i < numCols; i++)
				{
					tochange = LR * toAdjust[i] / numRows;
					parameters[i] -= tochange;
					toAdjust[i] = 0;
					if ((numIters % checkPoint == 0) && printRes)
					{
						printf("At iteration %d, parameters[% d] = % f\n", numIters, i, parameters[i]);
					}
				}
				numIters++;
			}
			if (printRes)
			{
				for (i = 0; i < numCols; i++)
				{
					printf("At iteration %d, parameters[% d] = % f\n", numIters, i, parameters[i]);
				}
			}
			printf("Time to do %d calculations with %d threads is: %f(ms)\n", maxIters, NT, totalMilliseconds);

			toAvgPrf[whichAmt] += totalMilliseconds;
			whichAmt++;
		}
		roundCounter++;
	}
	for (int tc = 0; tc < 9; tc++)
	{
		printf("Avg time to do %d calculations with %d threads over %d rounds is %f (ms)\n", maxIters, numThreadsArr[tc],numRounds, toAvgPrf[tc]/numRounds);
	}
			/*
	


	//hipDeviceSynchronize(stop);
	//auto stop = high_resolution_clock::now();
	//auto duration = duration_cast<microseconds>(stop - start);


	for (i = 0; i < 13; i++)
	{
		printf("c[% d] = % d\n", i, allCols[i]);
	}

	for (i = truSize - 10; i < truSize; i++)
	{
		printf("c[% d] = % d\n", i, allCols[i]);
	}
	hipFree(allCols);

	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
	*/
	/*
	1024 threads
	base = 0.65ms
	10x = 6.5ms
	20x = 13ms
	30x= 19.5

	512 threads
	base = 0.62ms
	10x = 6.02ms
	20x = 12.05ms
	30x = 18.22ms

	1 threads
	base = 18ms
	10x = 183ms
	20x = 354ms
	30x = 523ms

	2 threads

	base = 9ms
	30x = 263ms

	1024 threads, multiplication, 256 columns, 56.5ms
					addition (+), ... , 42.8
					addition (+=) 36.9
	*/


	return 0;
}


/*


const int maximumSize = 1024 * 1024 * 1.5;
const int parameterSize = 2;
const int numCols = parameterSize;
const int numRows = 10000;
const int numFiles = 168;

const int totalSize = 1024 * 1024 * 1;
const int NT = SIZE;
const int totalCols = 256;
const int truSize = numRows;
const float LR = 0.1;
const int maxIters = 2000;
const int checkPoint = 1000;

void read_record()
{

	// File pointer 
	std::fstream fin;

	// Open an existing file 
	int counter = 1;
	std::string pathName = "B:\\Github2\\Data512\\finalProject\\data\\pivotData\\block";
	std::string result, line, temp, word;
	std::vector<std::string> row;
	char* buffer = new char[3];
	while (counter < numFiles)
	{
		sprintf(buffer, "%d", counter);
		result = pathName + buffer + "sep.csv";
		fin.open(result, std::fstream::in);
		bool isgood = fin.good();
		while (fin >> temp) {

			row.clear();

			// used for breaking words 
			std::stringstream s(temp);

			// read every column data of a row and 
			// store it in a string variable, 'word' 
			while (std::getline(s, word, ',')) {
				printf("%s\n", word.c_str());
			}
		}
		fin.close();
		printf("%s\n", result.c_str());
		counter++;
	}
	std::free(buffer);

	//roll2 = stoi(row[0]);

}

__global__ void vectorAdd2(int* a, int* b, int* c, int n)
{
	//int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int iw = threadIdx.x;
	int j = blockDim.x;
	int z = blockIdx.x;
	int iy = threadIdx.y;
	int jy = blockDim.y;
	int zy = blockIdx.y;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
	{
		c[i] = (a[i] + b[i]) % 1024;
	}
}

//hipMallocManaged(&b, truSize * sizeof(int));
//hipMallocManaged(&c, truSize * sizeof(int));


__global__ void logRegression(float* outs, float* ins, float* theta, float* weightChanges)
{

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = 0;
	double total = 0.0;
	for (j = 0; j < numCols; j++)
	{
		total += theta[j] * ins[(numRows * j) + i];
	}
	float l = outs[i] - (1.0 / (1 + exp(-1 * total)));
	for (j = 0; j < numCols; j++)
	{
		float toadd = ((ins[(numRows * j) + i]) * l) / numRows;
		atomicAdd(&weightChanges[j], toadd);
	}

}

__global__ void vectorAdd(int* cols, int numCols, int n)
{
	//int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int iw = threadIdx.x;
	//int j = blockDim.x;
	//int z = blockIdx.x;
	//int iy = threadIdx.y;
	//int jy = blockDim.y;
	//int zy = blockIdx.y;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = 0;
	if (i < n)
	{
		for (j = 1; j < numCols; j++)
		{
			cols[i] += cols[j * truSize + i];
		}
	}
}

int main()
{

	//read_record();

	float* data = new float[numRows * numCols];
	float* output = new float[numRows];
	float* parameters = new float[numCols];
	float* toAdjust = new float[numCols];
	int i = 0;
	int j = 0;
	float insertVal = 0.0;
	float tochange = 0.0;
	int numIters = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMallocManaged(&parameters, sizeof(float) * numCols);
	hipMallocManaged(&toAdjust, sizeof(double) * numCols);
	hipMallocManaged(&output, sizeof(float) * numRows);
	hipMallocManaged(&data, sizeof(float) * numRows * numCols);

	dim3 numThreads(NT);
	dim3 gridDim((numRows * numCols / NT) + 1, 1);

	// to view array contents, do "arrayName,numView" e.g. data,10
	for (i = 0; i < numCols; i++)
	{
		if (i % 2 == 0)
		{
			insertVal = -0.2;
		}
		else
		{
			insertVal = 0.2;
		}
		for (j = 0; j < numRows; j++)
		{
			data[numRows * i + j] = insertVal;
		}
	}

	for (i = 0; i < numRows; i++)
	{
		output[i] = 1.0;
	}

	for (i = 0; i < numCols; i++)
	{
		parameters[i] = -0.01;
		toAdjust[i] = 0.0;
	}

	hipEventRecord(start);
	while (numIters < maxIters)
	{
		logRegression << <gridDim, numThreads >> > (output, data, parameters, toAdjust);


		for (i = 0; i < numCols; i++)
		{
			tochange = LR * toAdjust[i] / numRows;
			parameters[i] -= tochange;
			toAdjust[i] = 0;
			if (numIters % checkPoint == 0)
			{
				printf("parameters[% d] = % f\n", i, parameters[i]);
			}

		}
		numIters++;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time to %d iterations is: %f(ms)\n", maxIters, milliseconds);
	hipFree(parameters);
	hipFree(toAdjust);
	hipFree(output);
	hipFree(data);


	/*



//hipDeviceSynchronize(stop);
//auto stop = high_resolution_clock::now();
//auto duration = duration_cast<microseconds>(stop - start);


for (i = 0; i < 13; i++)
{
	printf("c[% d] = % d\n", i, allCols[i]);
}

for (i = truSize - 10; i < truSize; i++)
{
	printf("c[% d] = % d\n", i, allCols[i]);
}
hipFree(allCols);

int nDevices;

hipGetDeviceCount(&nDevices);
for (int i = 0; i < nDevices; i++) {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);
	printf("Device Number: %d\n", i);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n",
		prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n",
		prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
}
*/
/*
1024 threads
base = 0.65ms
10x = 6.5ms
20x = 13ms
30x= 19.5

512 threads
base = 0.62ms
10x = 6.02ms
20x = 12.05ms
30x = 18.22ms

1 threads
base = 18ms
10x = 183ms
20x = 354ms
30x = 523ms

2 threads

base = 9ms
30x = 263ms

1024 threads, multiplication, 256 columns, 56.5ms
				addition (+), ... , 42.8
				addition (+=) 36.9



	return 0;
}
*/